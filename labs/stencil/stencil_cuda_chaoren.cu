#include "hip/hip_runtime.h"
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <strings.h>
#include <string.h>
#include "bmp.h"
#include "omp.h"


extern "C" void LoadBMPFile(uchar3 **img, BMPHeader *hdr, BMPInfoHeader *infoHdr, const char *name);

extern "C" void WriteBMPFile(uchar3 **img, BMPHeader hdr, BMPInfoHeader infoHdr, const char *name);

#define idx(A,B) ((A) * cols + (B))
//#define new_idx(A, B, C) (A * strip_width * rows + B * strip_width + C)
//#define new_idx_last(A, B, C) (A * strip_width * rows + B * strip_width_last + C)

#define thrds 256

typedef struct pixel {
	float x, y, z;
} Pixel;

__device__ int indexFinder(int y, int x, int rowLen) {
  return y * rowLen + x;
}


__global__ void filter(int rows, int cols, Pixel *myimg, Pixel *oimg) 
{
  unsigned int tempThreadsx = threadsPerBlock.x + 2;
  unsigned int tempThreadsy = threadsPerBlock.y + 2;

  __shared__ Piexel* temp = cudamalloc(tempThreadsx * tempThreadsy *sizeof(Pixel));

  int globalx = blockIdx.x * threadsPerBlock.x + threadIdx.x;
  int globaly = blockIdx.y * threadsPerBlock.y + threadIdx.y;
  if (globalx < cols && globaly < rows) {
    usigned int gindex = indexFinder(globaly, globalx,  cols);
    usigned int tempx = threadIdx.x + 1;
    usigned int tempy = threadIdx.y + 1;
    usigned int bindex = indexFinder(tempy, tempx, tempThreadsx);
    temp[bindex] = myimg[gindex];   // load itself
    if (threadIdx.x == 0 && globalx != 0) {
      int leftbindex = bindex - 1;
      int leftgindex = gindex - 1;
      temp[leftbindex]  = myimg[leftgindex];
    }
    if (threadIdx.x == threadsPerBlock.x - 1 && globalx != cols - 1) {
 		int rightbindex = bindex + 1;
 		int rightgindex = gindex + 1;
 		temp[rightbindex] = myimg[rightgindex]
    }
    if(threadIdx.y == 0 && globaly != 0) {
    	int abovebindex = bindex - tempThreadsx;
    	int abovegindex = gindex - cols;
    	temp[abovebindex] = myimg[abovegindex];
    }
    if(threadIdx.y == threadsPerBlock.y - 1 && globaly ! = rows - 1) {
    	int underbindex = bindex + tempThreadsx;
    	int undergindex = gindex + cols;
    	temp[underbindex] = myimg[undergindex];
    }
    int cornerbindex, cornergindey;
    if (threadIdx.x == 0 and threadIdx.y == 0) { 
    	cornerbindex = bindex - tempThreadsx - 1;
    	cornergindey = gindex - cols - 1;
    	temp[cornerbindex] = myimg[cornergindex];
    }
     if (threadIdx.x == 0 and threadIdx.y == threadsPerBlock.y - 1) { 
    	cornerbindex = bindex + tempThreadsx - 1;
    	cornergindey = gindex + cols - 1;
    	temp[cornerbindex] = myimg[cornergindex];
    }
    if (threadIdx.x == threadsPerBlock.x - 1 and threadIdx.y == 0) { 
    	cornerbindex = bindex - tempThreadsx + 1;
    	cornergindey = gindex - cols + 1;
    	temp[cornerbindex] = myimg[cornergindex];
    }
    if (threadIdx.x == threadsPerBlock.x - 1 and threadIdx.y == threadsPerBlock.y - 1) { 
    	cornerbindex = bindex + tempThreadsx + 1;
    	cornergindey = gindex + cols + 1;
    	temp[cornerbindex] = myimg[cornergindex];
    }

    __syncthreads();
    if (globalx > 0 && globalx < cols - 1 && globaly > 0 and globaly < rows - 1) {
    	oimg[gindex].z = (temp[indexFinder(tempy, tempx, threadsPerBlock.x)].z 
    					+ temp[indexFinder(tempy, tempx-1, threadsPerBlock.x)].z
    					+ temp[indexFinder(tempy, tempx+1, threadsPerBlock.x)].z
    					+ temp[indexFinder(tempy-1, tempx, threadsPerBlock.x)].z
    					+ temp[indexFinder(tempy-1, tempx-1, threadsPerBlock.x)].z
    					+ temp[indexFinder(tempy-1, tempx+1, threadsPerBlock.x)].z
    					+ temp[indexFinder(tempy+1, tempx, threadsPerBlock.x)].z
    					+ temp[indexFinder(tempy+1, tempx-1, threadsPerBlock.x)].z
    					+ temp[indexFinder(tempy+1, tempx+1, threadsPerBlock.x)].z) / 9;

    	oimg[gindex].y = (temp[indexFinder(tempy, tempx, threadsPerBlock.x)].y 
    					+ temp[indexFinder(tempy, tempx-1, threadsPerBlock.x)].y
    					+ temp[indexFinder(tempy, tempx+1, threadsPerBlock.x)].y
    					+ temp[indexFinder(tempy-1, tempx, threadsPerBlock.x)].y
    					+ temp[indexFinder(tempy-1, tempx-1, threadsPerBlock.x)].y
    					+ temp[indexFinder(tempy-1, tempx+1, threadsPerBlock.x)].y
    					+ temp[indexFinder(tempy+1, tempx, threadsPerBlock.x)].y
    					+ temp[indexFinder(tempy+1, tempx-1, threadsPerBlock.x)].y
    					+ temp[indexFinder(tempy+1, tempx+1, threadsPerBlock.x)].y) / 9;

    	oimg[gindex].x = (temp[indexFinder(tempy, tempx, threadsPerBlock.x)].x 
    					+ temp[indexFinder(tempy, tempx-1, threadsPerBlock.x)].x
    					+ temp[indexFinder(tempy, tempx+1, threadsPerBlock.x)].x
    					+ temp[indexFinder(tempy-1, tempx, threadsPerBlock.x)].x
    					+ temp[indexFinder(tempy-1, tempx-1, threadsPerBlock.x)].x
    					+ temp[indexFinder(tempy-1, tempx+1, threadsPerBlock.x)].x
    					+ temp[indexFinder(tempy+1, tempx, threadsPerBlock.x)].x
    					+ temp[indexFinder(tempy+1, tempx-1, threadsPerBlock.x)].x
    					+ temp[indexFinder(tempy+1, tempx+1, threadsPerBlock.x)].x) / 9;
    }
  }
}

double  apply_stencil(const int rows, const int cols, Pixel * const in_d, Pixel * const out_d, Piexl * const out, uint64_t img_size) {
	dim3 threadsPerBlock(16, 16);
	blockx = cols % threadsPerBlock.x = 0 ? cols/threadsPerBlock.x : cols/threadsPerBlock.x + 1
	blocky = rows % threadsPerBlock.y = 0 ? rows/threadsPerBlock.y : rows/threadsPerBlock.y + 1
	dim3 numBlocks(blockx, blocky);
	double tstart, tend;
    tstart = omp_get_wtime();
	filter<<<numBlocks, threadsPerBlock>>>(rows, cols, in_d, out_d);
    tend = omp_get_wtime();
    hipMemcpy(out, out_d, img_size, hipMemcpyDeviceToHost);
	return(tend-tstart);
}

// main read, call filter, write new image
int main(int argc, char **argv)
{

  BMPHeader hdr;
  BMPInfoHeader infoHdr;
  uchar3 *bimg;
  Pixel *img,*oimg;
  Pixel *img_d, *oimg_d;
  uint64_t x,y;
//  uint64_t new_x, new_y, new_z;
  uint64_t img_size;
  double start, end;
//  int strip_width;

  if(argc != 2) {
    printf("Usage: %s imageName\n", argv[0]);
    return 1;
  }

  
  LoadBMPFile(&bimg, &hdr, &infoHdr, argv[1]);
  printf("Data init done: size = %d, width = %d, height = %d.\n",
	hdr.size, infoHdr.width, infoHdr.height);

  img_size = infoHdr.width * infoHdr.height * sizeof(Pixel);
  img = (Pixel *) malloc(img_size);
  hipMalloc((void **) &img_d, img_size);
  if (img == NULL) {
    printf("Error Cant alloc image space\n");
    exit(-1);
  }
  memset(img,0,img_size);
  oimg = (Pixel *) malloc(img_size);
  hipMalloc((void **) &oimg_d, img_size);
  if (oimg == NULL) {
    printf("Error Cant alloc output image space\n");
    exit(-1);
  }
  memset(oimg,0,img_size);
  printf("Convert image\n");
  // convert to floats for processing and data reorganization 
  int rows = infoHdr.height;
  int cols = infoHdr.width;
//  if (cols % strip_width != 0) { // the width of last strip is smaller than strip_width
//	int strip_num = cols / strip_width + 1
//	int strip_width_last = cols - (strip_num - 1) * strip_width
//  } else {
//	int strip_num = cols / strip_width
//	int strip_width_last = strip_width
//  }
	
  for (y=0; y<rows; y++)
    for (x=0; x<cols; x++)
    {
//	 new_z = x/step_width;
//	 new_y = y;
//	 new_x = x % step_width;
//	 img[idx(y,x)].x = bimg[idx(y,x)].x/255.0;   
//	 img[idx(y,x)].y = bimg[idx(y,x)].y/255.0;   
//	 img[idx(y,x)].z = bimg[idx(y,x)].z/255.0;   
	 img[idx(y,x)].x = bimg[idx(y,x)].x/255.0;   
	 img[idx(y,x)].y = bimg[idx(y,x)].y/255.0;   
	 img[idx(y,x)].z = bimg[idx(y,x)].z/255.0;   
    }   

// copy to cuda memory
  hipMemcpy(img_d, img, img_size, hipMemcpyHostToDevice);
  // hipMemcpy(oimg_d, oimg, img_size, hipMemcpyHostToDevice);
    
    double runtime;
    runtime = apply_stencil(infoHdr.height, infoHdr.width, img_d, oimg_d, oimg, img_size);
    printf("time for stencil = %f seconds\n",runtime);

  // clear bitmap array
  memset(bimg,0,infoHdr.height*infoHdr.width*3);
  double err = 0.0;
  // convert to uchar3 for output
printf("rows %d cols %d\n",rows, cols);
  for (y=0; y<rows; y++)
    for (x=0; x<cols; x++)
    {
	 bimg[idx(y,x)].x = oimg[idx(y,x)].x*255;   
	 bimg[idx(y,x)].y = oimg[idx(y,x)].y*255;   
	 bimg[idx(y,x)].z = oimg[idx(y,x)].z*255;   
         err += (img[idx(y,x)].x - oimg[idx(y,x)].x);
         err += (img[idx(y,x)].y - oimg[idx(y,x)].y);
         err += (img[idx(y,x)].z - oimg[idx(y,x)].z);
    }   
   printf("Cummulative error between images %g\n",err);

  // write the output file
  WriteBMPFile(&bimg, hdr,infoHdr, "./img-new.bmp");
  free(img); free(oimg); free(bimg); 
  hipFree(img_d); hipFree(oimg_d);
  
}
