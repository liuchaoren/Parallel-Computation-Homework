#include "hip/hip_runtime.h"
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <strings.h>
#include <string.h>
#include "bmp.h"
#include "omp.h"


extern "C" void LoadBMPFile(uchar3 **img, BMPHeader *hdr, BMPInfoHeader *infoHdr, const char *name);

extern "C" void WriteBMPFile(uchar3 **img, BMPHeader hdr, BMPInfoHeader infoHdr, const char *name);

#define idx(A,B) ((A) * cols + (B))
#define new_idx(A, B, C) (A * strip_width * rows + B * strip_width + C)
#define new_idx_last(A, B, C) (A * strip_width * rows + B * strip_width_last + C)

#define thrds 256
#define strip_width 16

typedef struct pixel {
	float x, y, z;
} Pixel;


__global__ void filter(Pixel *myimg, Pixel *oimg, int rows, int strip_width, int strip_width_last, int strip_num)
{
  __shared__ Piexel temp[(strip_width + 2)^2];
  int gindex = threadIdx.x + blockIdx.x * blockDim.x;
  int x = threadIdx.x 
  for (uint64_t y = 1; y < rows-1; y++)
    for (uint64_t x = 1; x < cols-1; x++) {
        oimg[idx(y,x)].z = 
          (myimg[idx(y,x)].z
           + myimg[idx(y,x-1)].z 
           + myimg[idx(y,x+1)].z 
           + myimg[idx(y-1,x)].z
           + myimg[idx(y-1,x-1)].z 
           + myimg[idx(y-1,x+1)].z 
           + myimg[idx(y+1,x)].z
           + myimg[idx(y+1,x-1)].z 
           + myimg[idx(y+1,x+1)].z)/9;

        oimg[idx(y,x)].y = 
         (myimg[idx(y,x)].y 
          + myimg[idx(y,x-1)].y 
          + myimg[idx(y,x+1)].y 
          + myimg[idx(y-1,x)].y 
          + myimg[idx(y-1,x-1)].y 
          + myimg[idx(y-1,x+1)].y 
          + myimg[idx(y+1,x)].y 
          + myimg[idx(y+1,x-1)].y 
          + myimg[idx(y+1,x+1)].y)/9;

        oimg[idx(y,x)].x = 
         (myimg[idx(y,x)].x 
          + myimg[idx(y,x-1)].x 
          + myimg[idx(y,x+1)].x 
          + myimg[idx(y-1,x)].x 
          + myimg[idx(y-1,x-1)].x 
          + myimg[idx(y-1,x+1)].x 
          + myimg[idx(y+1,x)].x 
          + myimg[idx(y+1,x-1)].x 
          + myimg[idx(y+1,x+1)].x)/9;
      }
}

double  apply_stencil(const int rows, const int cols, uint64_t img_size, int strip_width, int strip_width_last, int strip_num, Pixel * const in, Pixel * const out, Pixel * const in_d, Pixel * const out_d) {
    	hipMemcpy(in_d, in, img_size, hipMemcpyHostToDevice);
    	hipMemcpy(out_d, out, img_size, hipMemcpyHostToDevice);
	double tstart, tend;
      	tstart = omp_get_wtime();
	filter<<<(cols*rows + thrds - 1) / thrds, thrds>>>(in_d, out_d, rows, strip_width, strip_width_last, strip_num);
        tend = omp_get_wtime();
	return(tend-tstart);
}

// main read, call filter, write new image
int main(int argc, char **argv)
{

  BMPHeader hdr;
  BMPInfoHeader infoHdr;
  uchar3 *bimg;
  Pixel *img,*oimg;
  Pixel *img_d, *oimg_d;
  uint64_t x,y;
  uint64_t new_x, new_y, new_z;
  uint64_t img_size;
  double start, end;
//  int strip_width;

  if(argc != 2) {
    printf("Usage: %s imageName\n", argv[0]);
    return 1;
  }

  
  LoadBMPFile(&bimg, &hdr, &infoHdr, argv[1]);
  printf("Data init done: size = %d, width = %d, height = %d.\n",
	hdr.size, infoHdr.width, infoHdr.height);

  img_size = infoHdr.width * infoHdr.height * sizeof(Pixel);
  img = (Pixel *) malloc(img_size);
  hipMalloc((void **) &img_d, img_size);
  if (img == NULL) {
    printf("Error Cant alloc image space\n");
    exit(-1);
  }
  memset(img,0,img_size);
  oimg = (Pixel *) malloc(img_size);
  hipMalloc((void **) &oimg_d, img_size);
  if (oimg == NULL) {
    printf("Error Cant alloc output image space\n");
    exit(-1);
  }
  memset(oimg,0,img_size);
  printf("Convert image\n");
  // convert to floats for processing and data reorganization 
  int rows = infoHdr.height;
  int cols = infoHdr.width;
  if (cols % strip_width != 0) { // the width of last strip is smaller than strip_width
	int strip_num = cols / strip_width + 1
	int strip_width_last = cols - (strip_num - 1) * strip_width
  } else {
	int strip_num = cols / strip_width
	int strip_width_last = strip_width
  }
	
  for (y=0; y<rows; y++)
    for (x=0; x<cols; x++)
    {
	 new_z = x/step_width;
     	 new_y = y;
	 new_x = x % step_width;
//	 img[idx(y,x)].x = bimg[idx(y,x)].x/255.0;   
//	 img[idx(y,x)].y = bimg[idx(y,x)].y/255.0;   
//	 img[idx(y,x)].z = bimg[idx(y,x)].z/255.0;   
	 if (new_z < strip_num - 1) {
	 	img[new_idx(new_z, new_y, new_x)].x = bimg[idx(y,x)].x/255.0;   
	 	img[new_idx(new_z, new_y, new_x)].y = bimg[idx(y,x)].y/255.0;   
	 	img[new_idx(new_z, new_y, new_x)].z = bimg[idx(y,x)].z/255.0;   
	} else { 
	 	img[new_idx_last(new_z, new_y, new_x)].x = bimg[idx(y,x)].x/255.0;   
	 	img[new_idx_last(new_z, new_y, new_x)].y = bimg[idx(y,x)].y/255.0;   
	 	img[new_idx_last(new_z, new_y, new_x)].z = bimg[idx(y,x)].z/255.0;   
	}

    }   
    
    double runtime;
    runtime = apply_stencil(infoHdr.height, infoHdr.width, img_size, strip_width, strip_width_last, strip_num, img, oimg, img_d, oimg_d);
    printf("time for stencil = %f seconds\n",runtime);

  // clear bitmap array
  memset(bimg,0,infoHdr.height*infoHdr.width*3);
  double err = 0.0;
  // convert to uchar3 for output
printf("rows %d cols %d\n",rows, cols);
  for (y=0; y<rows; y++)
    for (x=0; x<cols; x++)
    {
	 bimg[idx(y,x)].x = oimg[idx(y,x)].x*255;   
	 bimg[idx(y,x)].y = oimg[idx(y,x)].y*255;   
	 bimg[idx(y,x)].z = oimg[idx(y,x)].z*255;   
         err += (img[idx(y,x)].x - oimg[idx(y,x)].x);
         err += (img[idx(y,x)].y - oimg[idx(y,x)].y);
         err += (img[idx(y,x)].z - oimg[idx(y,x)].z);
    }   
   printf("Cummulative error between images %g\n",err);

  // write the output file
  WriteBMPFile(&bimg, hdr,infoHdr, "./img-new.bmp");
  
}
