#include "hip/hip_runtime.h"
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <strings.h>
#include <string.h>
#include "bmp.h"
#include "omp.h"


extern "C" void LoadBMPFile(uchar3 **img, BMPHeader *hdr, BMPInfoHeader *infoHdr, const char *name);

extern "C" void WriteBMPFile(uchar3 **img, BMPHeader hdr, BMPInfoHeader infoHdr, const char *name);

#define idx(A,B) ((A) * cols + (B))

typedef struct pixel {
  float x, y, z;
} Pixel;

const int tile_size = 32;

__global__
void filter(Pixel *myimg, Pixel *oimg, int rows, int cols)
{
  __shared__ Pixel temp[tile_size+2][tile_size+2];

  int x = threadIdx.x + blockDim.x*blockIdx.x;
  int y = threadIdx.y + blockDim.y*blockIdx.y;

  int xx = threadIdx.x+1;
  int yy = threadIdx.y+1;

  int Dx = x+blockDim.x < cols ? blockDim.x : cols-x-1;
  int Dy = y+blockDim.y < rows ? blockDim.y : rows-y-1;
      	 
  // Copy pixels to shared memory
  if (x < cols && y < rows)
    {
      // Interior pixels
      temp[yy][xx] = myimg[idx(y,x)];
      // Left & right side pixels
      if(threadIdx.x == 0)
      	{
	  temp[yy][0] = myimg[idx(y,x-1)];
      	  temp[yy][Dx+1] = myimg[idx(y,x+Dx)];
      	}
      // Top & bottom pixels
      if(threadIdx.y == 0)
      	{
	  temp[0][xx] = myimg[idx(y-1,x)];
      	  temp[Dy+1][xx] = myimg[idx(y+Dy,x)];
      	}
      // Corner pixels
      if(threadIdx.x == 0 && threadIdx.y == 0)
      	{
      	  temp[0][0] = myimg[idx(y-1,x-1)];
      	  temp[0][Dx+1] = myimg[idx(y-1,x+Dx)];
      	  temp[Dy+1][0] = myimg[idx(y+Dy,x-1)];
      	  temp[Dy+1][Dx+1] = myimg[idx(y+Dy,x+Dx)];
      	}
    }

  __syncthreads();

  // Compute stencil for the block
  if (x > 0 && x < cols-1 && y > 0 && y < rows-1)
    {
      Pixel result;
      result.x = 0;
      result.y = 0;
      result.z = 0;

      for(int dy = -1; dy <=1; dy++)
	{
	  for(int dx = -1; dx <=1; dx++)
	    {
	      result.x += temp[yy+dy][xx+dx].x;
	      result.y += temp[yy+dy][xx+dx].y;
	      result.z += temp[yy+dy][xx+dx].z;
	    }
	}

      oimg[idx(y,x)].x = result.x/9;
      oimg[idx(y,x)].y = result.y/9;
      oimg[idx(y,x)].z = result.z/9;
    }
}

double  apply_stencil(const int rows, const int cols, Pixel * const in, Pixel * const out) {
  Pixel *d_in, *d_out;
  hipMalloc(&d_in, rows*cols*sizeof(Pixel));
  hipMalloc(&d_out, rows*cols*sizeof(Pixel));
  hipMemcpy(d_in, in, rows*cols*sizeof(Pixel), hipMemcpyHostToDevice);

  const dim3 blockSize(tile_size,tile_size,1);
  const dim3 gridSize((cols+tile_size-1)/tile_size,(rows+tile_size-1)/tile_size,1);

  double tstart, tend;
  tstart = omp_get_wtime();
  filter<<<gridSize, blockSize>>>(d_in, d_out, rows, cols);
  hipDeviceSynchronize();
  tend = omp_get_wtime();

  hipMemcpy(out, d_out, rows*cols*sizeof(Pixel), hipMemcpyDeviceToHost);
  return(tend-tstart);
}

// main read, call filter, write new image
int main(int argc, char **argv)
{

  BMPHeader hdr;
  BMPInfoHeader infoHdr;
  uchar3 *bimg;
  Pixel *img,*oimg;
  uint64_t x,y;
  uint64_t img_size;
  //double start, end;
  if(argc != 2) {
    printf("Usage: %s imageName\n", argv[0]);
    return 1;
  }

  
  LoadBMPFile(&bimg, &hdr, &infoHdr, argv[1]);
  printf("Data init done: size = %d, width = %d, height = %d.\n",
	hdr.size, infoHdr.width, infoHdr.height);

  img_size = infoHdr.width * infoHdr.height * sizeof(Pixel);
  img = (Pixel *) malloc(img_size);
  if (img == NULL) {
    printf("Error Cant alloc image space\n");
    exit(-1);
  }
  memset(img,0,img_size);
  oimg = (Pixel *) malloc(img_size);
  if (oimg == NULL) {
    printf("Error Cant alloc output image space\n");
    exit(-1);
  }
  memset(oimg,0,img_size);
  printf("Convert image\n");
  // convert to floats for processing
  int rows = infoHdr.height;
  int cols = infoHdr.width;
  for (y=0; y<rows; y++)
    for (x=0; x<cols; x++)
    {
	 img[idx(y,x)].x = bimg[idx(y,x)].x/255.0;   
	 img[idx(y,x)].y = bimg[idx(y,x)].y/255.0;   
	 img[idx(y,x)].z = bimg[idx(y,x)].z/255.0;   
    }   

    double runtime;
    runtime = apply_stencil(infoHdr.height, infoHdr.width, img, oimg);
    printf("time for stencil = %f seconds\n",runtime);

  // clear bitmap array
  memset(bimg,0,infoHdr.height*infoHdr.width*3);
  double err = 0.0;
  // convert to uchar3 for output
printf("rows %d cols %d\n",rows, cols);
  for (y=0; y<rows; y++)
    for (x=0; x<cols; x++)
    {
	 bimg[idx(y,x)].x = oimg[idx(y,x)].x*255;   
	 bimg[idx(y,x)].y = oimg[idx(y,x)].y*255;   
	 bimg[idx(y,x)].z = oimg[idx(y,x)].z*255;   
         err += (img[idx(y,x)].x - oimg[idx(y,x)].x);
         err += (img[idx(y,x)].y - oimg[idx(y,x)].y);
         err += (img[idx(y,x)].z - oimg[idx(y,x)].z);
    }   
   printf("Cummulative error between images %g\n",err);

  // write the output file
  WriteBMPFile(&bimg, hdr,infoHdr, "./img-new.bmp");
  
}
